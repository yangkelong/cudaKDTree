#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "cukd/builder.h"
#include "cukd/fcp.h"  // fcp = "find closest point" query
#include <queue>
#include <iomanip>
#include <random>

//using mydata3 = float3;
using mydata3 = double3;
using mydata = typename cukd::scalar_type_of<mydata3>::type;

template<typename T>
T *generatePoints(int N){
  static int g_seed = 100000;
  std::seed_seq seq{g_seed++};
  std::default_random_engine rd(seq);
  std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
  //std::uniform_int_distribution<> dist(0, N);
  std::uniform_real_distribution<> dist(-1, 1);
  std::cout << "generating " << N << " uniform random points" << std::endl;
  T *d_points = 0;
  hipMallocManaged((char **)&d_points, N * sizeof(*d_points));
  if (!d_points)
    throw std::runtime_error("could not allocate points mem...");

  for (int i = 0; i < N; i++){
    d_points[i].x = (mydata)dist(gen);
    d_points[i].y = (mydata)dist(gen);
    d_points[i].z = (mydata)dist(gen);
  }
  return d_points;
}


__global__ void d_fcp(mydata *d_results,
                      mydata3 *d_queries,
                      int numQueries,
                      /*! the world bounding box computed by the builder */
                      const cukd::box_t<mydata3> *d_bounds,
                      mydata3 *d_nodes,
                      int numNodes,
                      mydata cutOffRadius){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numQueries)
    return;

  mydata3 queryPos = d_queries[tid];
  cukd::FcpSearchParams params;
  params.cutOffRadius = cutOffRadius;
  int closestID = cukd::cct::fcp(queryPos, *d_bounds, d_nodes, numNodes, params);
  d_results[tid] = (closestID < 0)
                       ? INFINITY
                       : cukd::distance(queryPos, d_nodes[closestID]);
}

int main(int ac, const char **av){
  // using namespace cukd::common;

  int numPoints = 1000000;
  int nRepeats = 1;
  size_t numQueries = 1000000;
  // 搜索时的最大半径
  mydata cutOffRadius = std::numeric_limits<mydata>::infinity();

  for (int i = 1; i < ac; i++)
  {
    std::string arg = av[i];
    if (arg[0] != '-')
      numPoints = std::stoi(arg);
    else if (arg == "-nq")
      numQueries = atoi(av[++i]);
    else if (arg == "-nr")
      nRepeats = atoi(av[++i]);
    else if (arg == "-r")
      cutOffRadius = std::stof(av[++i]);
    else
      throw std::runtime_error("known cmdline arg" + arg);
  }

  // ==================================================================
  // create sample input point that we'll build the tree over
  // ==================================================================


  mydata3 *d_points = generatePoints<mydata3>(numPoints);

  // ==================================================================
  // allocate some memory for the world-space bounding box, so the
  // builder can compute and return that for our chosen traversal
  // method to use
  // ==================================================================
  cukd::box_t<mydata3> *d_bounds;
  hipMallocManaged((void **)&d_bounds, sizeof(cukd::box_t<mydata3>));
  std::cout << "allocated memory for the world space bounding box ..." << std::endl;

  // ==================================================================
  // build the tree. this will also comptue the world-space boudig box
  // of all points
  // ==================================================================
  std::cout << "calling builder..." << std::endl;
  double t0 = cukd::common::getCurrentTime();
  cukd::buildTree(d_points, numPoints, d_bounds);
  CUKD_CUDA_SYNC_CHECK();
  double t1 = cukd::common::getCurrentTime();
  std::cout << "done building tree, took "
            << cukd::common::prettyDouble(t1 - t0) << "s" << std::endl;

  // ==================================================================
  // create set of sample query points
  // ==================================================================
  mydata3 *d_queries = generatePoints<mydata3>(numQueries);
  // allocate memory for the results
  mydata *d_results;
  CUKD_CUDA_CALL(MallocManaged((void **)&d_results, numQueries * sizeof(*d_results)));

  // ==================================================================
  // and do some queryies - let's do the same ones in a loop so we cna
  // measure perf.
  // ==================================================================
  {
    double t0 = cukd::common::getCurrentTime();
    for (int i = 0; i < nRepeats; i++)
    {
      int bs = 128;
      int nb = cukd::divRoundUp((int)numQueries, bs);
      d_fcp<<<nb, bs>>>(d_results, d_queries, numQueries,
                        d_bounds, d_points, numPoints, cutOffRadius);
      hipDeviceSynchronize();
    }
    CUKD_CUDA_SYNC_CHECK();
    double t1 = cukd::common::getCurrentTime();
    std::cout << "done " << nRepeats
              << " iterations of " << numQueries
              << " fcp queries, took " << cukd::common::prettyDouble(t1 - t0)
              << "s" << std::endl;
    std::cout << "that is " << cukd::common::prettyDouble(numQueries * nRepeats / (t1 - t0))
              << " queries/s" << std::endl;
  }
}
